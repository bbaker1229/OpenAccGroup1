
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <unistd.h>

#define BSIZE  16
#define NN 4000;
#define MM 6000;

__global__ void MatAdd(int N, int M, float *A, float *B, float *C){ 
   int j = blockIdx.x * blockDim.x + threadIdx.x;
   int i = blockIdx.y * blockDim.y + threadIdx.y;
   if (i < M && j < N)
       C[i*N+j] = A[i*N+j] + B[i*N+j]; 
} 

 void err_exit(char *message);
 float mat_add_check(int n,  float *x, float *y, float *z)  {
 float s=0.0, t = 0.0, td = 0.0;
 for (int i=0; i<n; i++) {
       s  = y[i]+x[i]-z[i]; 
       t += s*s ;
       td += (x[i]*x[i]+y[i]*y[i]);
 }    

//-------------------- matrices are both zero
 if (td == 0.0) return(-1);
    else
//-------------------- normal return
   return(sqrt(s/td));
} 

double wctime() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return (tv.tv_sec + 1E-6 * tv.tv_usec);
}

int main() {
float *Ad, *Bd, *Cd; 
float  *A,  *B,  *C; 
int N, M, i, j; 
size_t MatSize;
float s;
double wctime();
double t1;
float nops;
//-------------------- set dimension N
 N = NN;
 M = MM;

 char LineG[] = "Error allocating GPU  memory";
 char LineH[] = "Error allocating Host memory";

  
 MatSize = N*M*sizeof(float);
//-------------------- allocate on cpu
 A = (float *)malloc(MatSize);        
 B = (float *)malloc(MatSize);        
 C = (float *)malloc(MatSize);    
 if ((A==NULL) | (B==NULL) | (C==NULL) ) 
          err_exit(LineH);
//-------------------- allocate on GPU
 if (hipMalloc((void **) &Ad, MatSize) != hipSuccess) 
       err_exit(LineG);
 if (hipMalloc((void **) &Bd, MatSize) != hipSuccess) 
       err_exit(LineG);
 if (hipMalloc((void **) &Cd, MatSize) != hipSuccess) 
       err_exit(LineG);
//-------------------- fill arrays A,B

 for (i=0; i<M; i++) 
    for (j=0; j<N; j++) {
      A[i*N+j] = (float) rand() / (float) rand();
      B[i*N+j] = (float) rand() / (float) rand();
} 
//
//-------------------- copy matrices A,B+ to GPU memory
t1 = wctime();
hipMemcpy(Ad, A, MatSize, hipMemcpyHostToDevice);
hipMemcpy(Bd, B, MatSize, hipMemcpyHostToDevice);
//-------------------- Kernel invocation
   dim3 dimBlock(BSIZE, 256/BSIZE);
// x: columns , y: rows    
   dim3 dimGrid((N + dimBlock.x-1) / dimBlock.x,
                (M + dimBlock.y-1) / dimBlock.y);
   MatAdd<<<dimGrid, dimBlock>>>(N, M, Ad, Bd, Cd);
//-------------------- see if things did execute 
 hipError_t error = hipGetLastError();
 if (error) {
     printf("CUDA error: %s \n",hipGetErrorString(error));
     exit(1);
 }
//-------------------- Transfer result from GPU to CPU
hipMemcpy(C, Cd, MatSize, hipMemcpyDeviceToHost);
t1 = (wctime() - t1);
//-------------------- check whether addition was correct
s =  mat_add_check(N*M,A,B,C);
 
printf(" Mat dims M = %d  N = %d  -- err= %10.6e\n",M,N,s); 
printf(" Function runtime = %f seconds\n",t1);
t1 = t1 * 1.e+06;
nops = (float) M*N;
printf(" Performance = %f Mflops\n",nops/t1);
//-------------------- Free Host arrays
 free(A); 
 free(B);
 free(C);
//-------------------- Free GPU memory
 hipFree(Ad);
 hipFree(Bd);
 hipFree(Cd);	
}

//-------------------- Prints error error Msg and exits 
void err_exit(char *errMsg) {
	printf("%s\n", errMsg);
	exit(1);
}
